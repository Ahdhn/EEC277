#include "hip/hip_runtime.h"
//Perform two float vector dot product on gpu and cpu and produce the average absolute error 
//By: Ahmed H. Mahmoud

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <conio.h>

__global__ void
DorProduct(const double *d_vect1, const double *d_vect2, int len, double*d_res)
{
	//we just launch one register and let it do all the work through a loop

    //int idx = blockDim.x * blockIdx.x + threadIdx.x;	
	//printf("\n idx =%d", idx);
	d_res[0] = 0;
	for (size_t i = 0; i < len; i++){		
		d_res[0] += d_vect1[i] * d_vect2[i];
	}
}


int main(void)
{
	// set the device to my GT 610
	int deviceCount;
	hipGetDeviceCount(&deviceCount);
	int dev = 1;
	hipSetDevice(dev);	
	    
	//length of vectors 
	int len = 100000;	
	size_t size = len * sizeof(double);

    //allocate stuff on the host     
	double*h_vect1 = new double[len];
	double*h_vect2 = new double[len];
	double*h_res = new double[1];
	
	srand ( time(NULL) ); // activate for different experiments
    //initialize stuff on the host
	for (int i = 0; i < len; ++i){ 
		h_vect1[i] = double(rand() / (double)RAND_MAX);
		h_vect2[i] = double(rand() / (double)RAND_MAX);
    }
	h_res[0] = 0;
	

	// Error code to check return values for CUDA calls
	hipError_t err = hipSuccess;


	//allocate stuff on the device     
	double *d_vect1 = NULL;
    err = hipMalloc((void **)&d_vect1, size);
    if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device d_vect1 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
		
	double *d_vect2 = NULL;
    err = hipMalloc((void **)&d_vect2, size);
	if (err != hipSuccess){ 
        fprintf(stderr, "Failed to allocate device d_vect2 (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	// Allocate result on GPU
	double*d_res = NULL;
	err = hipMalloc((void **)&d_res, sizeof(double));
	if (err != hipSuccess){
		fprintf(stderr, "Failed to allocate device d_res (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	   
    // Copy the from host memory to the device memory 
    err = hipMemcpy(d_vect1, h_vect1, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy h_vect1 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err = hipMemcpy(d_vect2, h_vect2, size, hipMemcpyHostToDevice);
    if (err != hipSuccess){
        fprintf(stderr, "Failed to copy h_vect2 from host to device (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

	err = hipMemcpy(d_res, h_res, sizeof(double), hipMemcpyHostToDevice);//useless but meh
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy h_res from host to device (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    //launch CUDA Kernel        
	dim3 blockSize(1,1,1), gridSize(1,1,1);	
	DorProduct <<<1, 1 >>>(d_vect1, d_vect2, len, d_res);
    err = hipGetLastError();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
	hipDeviceSynchronize();
		
	
	err = hipMemcpy(h_res, d_res, sizeof(double), hipMemcpyDeviceToHost);
	if (err != hipSuccess){
		fprintf(stderr, "Failed to copy d_res from device to host (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    //get cpu results 	
	double *inter_vect = new double[len];
	//make sure it is not converted into mulitply-add 
	double cpu_res(0);
	for (int i = 0; i < len; ++i){ 
		inter_vect[i] = h_vect1[i] * h_vect2[i];	
		//cpu_res += h_vect1[i] * h_vect2[i];
    }
	
	for (int i = 0; i < len; ++i){
		cpu_res += inter_vect[i];
	}    

    // Free device global memory
    err = hipFree(d_vect1);
    if (err != hipSuccess){
		fprintf(stderr, "Failed to free device d_vect1 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}

    err = hipFree(d_vect2);
    if (err != hipSuccess){
		fprintf(stderr, "Failed to free device d_vect2 (error code %s)!\n", hipGetErrorString(err));
		exit(EXIT_FAILURE);
	}
	    
    // Free host memory
    free(h_vect1);
    free(h_vect2);    
	//free(inter_vect);
	    
    err = hipDeviceReset();

    if (err != hipSuccess){
        fprintf(stderr, "Failed to deinitialize the device! error=%s\n", hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


	/// cal the error
	double error_cpu_gpu = abs(cpu_res - h_res[0]);




    printf("\nDone\n");

	printf("\n\nPress Any Key To Exit!!!");
	_getch();

    return 0;
}

